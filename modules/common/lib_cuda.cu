#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/core/core.hpp>
#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>
#include <opencv2/core/cuda/common.hpp>

__global__ void cuda_compression_kernel(const uchar3* input, uchar3* output, int width, int height, int reduction_factor) {
    int new_height = height / reduction_factor;
    int new_width  = width  / reduction_factor;

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < new_width && y < new_height) {
        int original_x = x * reduction_factor;
        int original_y = y * reduction_factor;

        output[y * new_width + x] = input[original_y * width + original_x];
    }
}

cv::Mat cuda_compression_algorithm(cv::Mat& frame, const int reduction_factor) {
    int width = frame.cols;
    int height = frame.rows;

    int new_height = height / reduction_factor;
    int new_width = width / reduction_factor;

    cv::Mat compressed_frame(new_height, new_width, frame.type());

    const auto in_bytes = frame.step * frame.rows;
    const auto out_bytes = compressed_frame.step * compressed_frame.rows;

    // Allocate device memory
    uchar3* d_input;
    uchar3* d_output;
    hipMalloc((void**)&d_input, in_bytes);
    hipMalloc((void**)&d_output, out_bytes);

    // Copy input data to device memory
    hipMemcpy(d_input, frame.ptr<uchar3>(), width * height * sizeof(uchar3), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    const dim3 block(16,16);

    // Calculate grid size to cover the whole image
    const dim3 grid(cv::cuda::device::divUp(frame.cols, block.x), cv::cuda::device::divUp(frame.rows, block.y));

    // Launch the CUDA kernel
    cuda_compression_kernel<<<grid, block>>>(d_input, d_output, width, height, reduction_factor);

    // Synchronize to check for any kernel launch errors
    cudaSafeCall( hipDeviceSynchronize() );

    // Copy output data from device memory
    cudaSafeCall(
            hipMemcpy(
                    compressed_frame.ptr<uchar3>(),
                    d_output, new_width * new_height * sizeof(uchar3),
                    hipMemcpyDeviceToHost)
    );

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);

    return compressed_frame;
}


